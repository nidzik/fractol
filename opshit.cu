/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   opshit.cu                                          :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: nidzik <nidzik@student.42.fr>              +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2015/04/29 11:22:56 by nidzik            #+#    #+#             */
/*   Updated: 2015/04/29 11:35:38 by nidzik           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include <hip/hip_runtime.h>
#include <cuda_call.h>
#include <stdio.h>

extern "C" void ft_set_pixel(t_benv *be)
{
  t_env e;
  e = ft_init_env(*be);
  e.x = 0;
  e.y = 0;
  while (e.x < e.image_x)
    {
      e.y = 0;
      while (e.y < e.image_y)
	{
		e.c_r = 1.5 * (e.x - 400) /	(be->factor * 400) + be->movex;
		e.c_i = 1.5 * (e.y - 400) /	(be->factor * 400) + be->movey;
		e.z_r = 0;
		e.z_i = 0;
		e.i = 0;
	    while ((e.z_r * e.z_r + e.z_i * e.z_i < 4 && e.i < e.ite_max) || e.i == 0)
	    {
	      e.tmp = e.z_r;
	      e.z_r = e.z_r * e.z_r - e.z_i * e.z_i + e.c_r;
	      e.z_i = 2 * e.z_i * e.tmp + e.c_i;
	      e.i ++;
	    }
	    if (e.i == e.ite_max && (e.y * be->size_line + 4 * e.x) < (be->size_line * w_wind))
	      be->data[e.y * be->size_line + 4 * e.x] = mlx_get_color_value(be->mlx, 0xff0000);
	    else
	      be->data[e.y * be->size_line + 4 * e.x] = mlx_get_color_value(be->mlx, 0x0000ff);
	    e.y+=3;
	}
      e.x+=3;
    } 
  mlx_put_image_to_window(be->mlx, be->win, be->img, 0, 0);

}
